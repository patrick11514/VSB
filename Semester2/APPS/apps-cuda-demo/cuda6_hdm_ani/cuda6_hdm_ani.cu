#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"
#include "animation.h"

// Demo kernel to create chess board
__global__ void kernel_creategradient( CudaImg t_color_cuda_img )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    int l_dy = l_x * t_color_cuda_img.m_size.y / t_color_cuda_img.m_size.x + l_y - t_color_cuda_img.m_size.y;
    unsigned char l_color = 255 * abs( l_dy ) / t_color_cuda_img.m_size.y;

    uchar3 l_bgr = ( l_dy < 0 ) ? ( uchar3 ) { l_color, 255 - l_color, 0 } : ( uchar3 ) { 0, 255 - l_color, l_color };

    // Store point into image
    t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ] = l_bgr;
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaImg t_big_cuda_img, CudaImg t_small_cuda_img, int2 t_position )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_small_cuda_img.m_size.y ) return;
    if ( l_x >= t_small_cuda_img.m_size.x ) return;
    int l_by = l_y + t_position.y;
    int l_bx = l_x + t_position.x;
    if ( l_by >= t_big_cuda_img.m_size.y || l_by < 0 ) return;
    if ( l_bx >= t_big_cuda_img.m_size.x || l_bx < 0 ) return;

    // Get point from small image
    uchar4 l_fg_bgra = t_small_cuda_img.m_p_uchar4[ l_y * t_small_cuda_img.m_size.x + l_x ];
    uchar3 l_bg_bgr = t_big_cuda_img.m_p_uchar3[ l_by * t_big_cuda_img.m_size.x + l_bx ];
    uchar3 l_bgr = { 0, 0, 0 };

    // compose point from small and big image according alpha channel
    l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
    l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
    l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

    // Store point into image
    t_big_cuda_img.m_p_uchar3[ l_by * t_big_cuda_img.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaImg t_big_cuda_img, CudaImg t_small_cuda_img, int2 t_position )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 32;
    dim3 l_blocks( ( t_small_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( t_small_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_insertimage<<< l_blocks, l_threads >>>( t_big_cuda_img, t_small_cuda_img, t_position );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

void Animation::start( CudaImg t_bg_cuda_img, CudaImg t_ins_cuda_img )
{
    if ( m_initialized ) return;
    hipError_t l_cerr;

    m_bg_cuda_img = t_bg_cuda_img;
    m_res_cuda_img = t_bg_cuda_img;
    m_ins_cuda_img = t_ins_cuda_img;

    // Memory allocation in GPU device
    // Memory for background
    l_cerr = hipMalloc( &m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof( uchar3 ) );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Memory for inserted image
    l_cerr = hipMalloc( &m_ins_cuda_img.m_p_void, m_ins_cuda_img.m_size.x * m_ins_cuda_img.m_size.y * sizeof( uchar4 ) );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Memory for result
    l_cerr = hipMalloc( &m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof( uchar3 ) );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Creation of background gradient
    int l_block_size = 32;
    dim3 l_blocks( ( m_bg_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( m_bg_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_creategradient<<< l_blocks, l_threads >>>( m_bg_cuda_img );

    // Copy data to GPU device
    l_cerr = hipMemcpy( m_ins_cuda_img.m_p_void, t_ins_cuda_img.m_p_void, m_ins_cuda_img.m_size.x * m_ins_cuda_img.m_size.y * sizeof( uchar4 ), hipMemcpyHostToDevice );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    m_initialized = 1;
}

void Animation::next( CudaImg t_res_pic, int2 t_position )
{
    if ( !m_initialized ) return;

    hipError_t cerr;

    // Copy data internally GPU from background into result
    cerr = hipMemcpy( m_res_cuda_img.m_p_void, m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToDevice );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

    // insert picture
    int l_block_size = 32;
    dim3 l_blocks( ( m_ins_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( m_ins_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_insertimage<<< l_blocks, l_threads >>>( m_res_cuda_img, m_ins_cuda_img, t_position );

    // Copy data to GPU device
    cerr = hipMemcpy( t_res_pic.m_p_void, m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

}

void Animation::stop()
{
    if ( !m_initialized ) return;

    hipFree( m_bg_cuda_img.m_p_void );
    hipFree( m_res_cuda_img.m_p_void );
    hipFree( m_ins_cuda_img.m_p_void );

    m_initialized = 0;
}


