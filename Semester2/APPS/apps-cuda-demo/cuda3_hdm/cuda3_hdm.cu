#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Manipulation with prepared image.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Every threads identifies its position in grid and in block and modify image
__global__ void kernel_animation( CudaImg t_cuda_img )
{
    // X,Y coordinates 
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_x >= t_cuda_img.m_size.x ) return;
    if ( l_y >= t_cuda_img.m_size.y ) return;

    // Point [l_x,l_y] selection from image
    uchar3 l_bgr, l_tmp = t_cuda_img.m_p_uchar3[ l_y * t_cuda_img.m_size.x + l_x ];

    // color rotation
    l_bgr.x = l_tmp.y;
    l_bgr.y = l_tmp.z;
    l_bgr.z = l_tmp.x;

    // Store point [l_x,l_y] back to image
    t_cuda_img.m_p_uchar3[ l_y * t_cuda_img.m_size.x + l_x ] = l_bgr;
}

void cu_run_animation( CudaImg t_img, uint2 t_block_size )
{
    hipError_t l_cerr;

    CudaImg l_cuda_pic;
    l_cuda_pic.m_size = t_img.m_size;

    // Memory allocation in GPU device
    l_cerr = hipMalloc( &l_cuda_pic.m_p_void, l_cuda_pic.m_size.x * l_cuda_pic.m_size.y * sizeof( uchar3 ) );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Copy data to GPU device
    l_cerr = hipMemcpy( l_cuda_pic.m_p_void, t_img.m_p_void, l_cuda_pic.m_size.x * l_cuda_pic.m_size.y * sizeof( uchar3 ), hipMemcpyHostToDevice );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Grid creation with computed organization
    dim3 l_grid( ( l_cuda_pic.m_size.x + t_block_size.x - 1 ) / t_block_size.x,
                 ( l_cuda_pic.m_size.y + t_block_size.y - 1 ) / t_block_size.y );
    kernel_animation<<< l_grid, dim3( t_block_size.x, t_block_size.y ) >>>( l_cuda_pic );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Copy data from GPU device to PC
    l_cerr = hipMemcpy( t_img.m_p_void, l_cuda_pic.m_p_void, t_img.m_size.x * t_img.m_size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Free memory
    hipFree( l_cuda_pic.m_p_void );

    hipDeviceSynchronize();
}
