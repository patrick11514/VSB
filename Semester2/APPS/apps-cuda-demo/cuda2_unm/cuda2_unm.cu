/// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology usage with unified memory.
//
// Multiplication of elements in float array.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

// Demo kernel for array elements multiplication.
// Every thread selects one element and multiply it. 
__global__ void kernel_mult( float *t_array, int t_length, float t_mult )
{
    int inx = blockDim.x * blockIdx.x + threadIdx.x;
    // if grid is greater then length of array...
    if ( inx >= t_length ) return;

    t_array[ inx ] *= t_mult;
}

void cu_run_mult( float *t_array, int t_length, float t_mult )
{
    hipError_t l_cerr;
    int l_threads = 128;
    int l_blocks = ( t_length + l_threads - 1 ) / l_threads;

    // Grid creation
    kernel_mult<<< l_blocks, l_threads >>>( t_array, t_length, t_mult );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}
