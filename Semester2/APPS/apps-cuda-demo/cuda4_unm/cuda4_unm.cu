#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale( CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    // Get point from color picture
    uchar3 l_bgr = t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ];

    // Store BW point to new image
    t_bw_cuda_img.m_p_uchar1[ l_y * t_bw_cuda_img.m_size.x + l_x ].x = l_bgr.x * 0.11 + l_bgr.y * 0.59 + l_bgr.z * 0.30;
}

void cu_run_grayscale( CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size, ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_grayscale<<< l_blocks, l_threads >>>( t_color_cuda_img, t_bw_cuda_img );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}
