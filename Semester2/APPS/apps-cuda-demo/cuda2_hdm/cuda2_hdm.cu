// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology usage without unified memory.
//
// Multiplication of elements in float array.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

// Demo kernel for array elements multiplication.
// Every thread selects one element and multiply it. 
__global__ void kernel_mult( float *t_array, int t_length, float t_mult )
{
    int l_inx = blockDim.x * blockIdx.x + threadIdx.x;
    // if grid is greater then length of array...
    if ( l_inx >= t_length ) return;

    t_array[ l_inx ] *= t_mult;
}

void cu_run_mult( float *t_array, int t_length, float t_mult )
{
    hipError_t l_cerr;
    int l_threads = 128;
    int l_blocks = ( t_length + l_threads - 1 ) / l_threads;

    // Memory allocation in GPU device
    float *l_cuda_array;
    l_cerr = hipMalloc( &l_cuda_array, t_length * sizeof( float ) );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Copy data from PC to GPU device
    l_cerr = hipMemcpy( l_cuda_array, t_array, t_length * sizeof( float ), hipMemcpyHostToDevice );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Grid creation
    kernel_mult<<< l_blocks, l_threads >>>( l_cuda_array, t_length, t_mult );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Copy data from GPU device to PC
    l_cerr = hipMemcpy( t_array, l_cuda_array, t_length * sizeof( float ), hipMemcpyDeviceToHost );
    if ( l_cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Free memory
    hipFree( l_cuda_array );
}
