#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Demo kernel to create chess board
__global__ void kernel_chessboard( CudaImg t_color_cuda_img )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    unsigned char b_or_w = 255 * ( ( blockIdx.x + blockIdx.y ) & 1 );

    // Store point into image
    t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ] = { b_or_w, b_or_w, b_or_w };
}

void cu_create_chessboard( CudaImg t_color_cuda_img, int t_square_size )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + t_square_size - 1 ) / t_square_size,
                   ( t_color_cuda_img.m_size.y + t_square_size - 1 ) / t_square_size );
    dim3 l_threads( t_square_size, t_square_size );
    kernel_chessboard<<< l_blocks, l_threads >>>( t_color_cuda_img );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_alphaimg( CudaImg t_color_cuda_img, uchar3 t_color )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    int l_diagonal = sqrtf( t_color_cuda_img.m_size.x * t_color_cuda_img.m_size.x + t_color_cuda_img.m_size.y * t_color_cuda_img.m_size.y );
    int l_dx = l_x - t_color_cuda_img.m_size.x / 2;
    int l_dy = l_y - t_color_cuda_img.m_size.y / 2;
    int l_dxy = sqrtf( l_dx * l_dx + l_dy * l_dy ) - l_diagonal / 2;

    // Store point into image
    t_color_cuda_img.m_p_uchar4[ l_y * t_color_cuda_img.m_size.x + l_x ] =
        { t_color.x, t_color.y, t_color.z, ( unsigned char ) ( 255 - 255 * l_dxy / ( l_diagonal / 2 ) ) };
}

void cu_create_alphaimg( CudaImg t_color_cuda_img, uchar3 t_color )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 32;
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_alphaimg<<< l_blocks, l_threads >>>( t_color_cuda_img, t_color );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaImg t_big_cuda_pic, CudaImg t_small_cuda_pic, int2 t_position )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_small_cuda_pic.m_size.y ) return;
    if ( l_x >= t_small_cuda_pic.m_size.x ) return;
    int l_by = l_y + t_position.y;
    int l_bx = l_x + t_position.x;
    if ( l_by >= t_big_cuda_pic.m_size.y || l_by < 0 ) return;
    if ( l_bx >= t_big_cuda_pic.m_size.x || l_bx < 0 ) return;

    // Get point from small image
    uchar4 l_fg_bgra = t_small_cuda_pic.m_p_uchar4[ l_y * t_small_cuda_pic.m_size.x + l_x ];
    uchar3 l_bg_bgr = t_big_cuda_pic.m_p_uchar3[ l_by * t_big_cuda_pic.m_size.x + l_bx ];
    uchar3 l_bgr = { 0, 0, 0 };

    // compose point from small and big image according alpha channel
    l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
    l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
    l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

    // Store point into image
    t_big_cuda_pic.m_p_uchar3[ l_by * t_big_cuda_pic.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaImg t_big_cuda_pic, CudaImg t_small_cuda_pic, int2 t_position )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 32;
    dim3 l_blocks( ( t_small_cuda_pic.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( t_small_cuda_pic.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_insertimage<<< l_blocks, l_threads >>>( t_big_cuda_pic, t_small_cuda_pic, t_position );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}
