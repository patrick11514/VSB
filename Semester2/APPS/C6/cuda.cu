#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

__global__ void kernel_overlap( CudaImg one, CudaImg two, int x, int y )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= one.m_size.y ) return;
    if ( l_x >= one.m_size.x ) return;

    if (l_x < x || l_y < y || l_x > (x + two.m_size.x) || l_y > (y + two.m_size.y)) return;

    if (two.m_p_uchar4[(l_y - y) * two.m_size.x + (l_x - x)].w == 0) return;

    one.m_p_uchar3[l_y * one.m_size.x + l_x].x = two.m_p_uchar4[(l_y - y) * two.m_size.x + (l_x - x)].x;
    one.m_p_uchar3[l_y * one.m_size.x + l_x].y = two.m_p_uchar4[(l_y - y) * two.m_size.x + (l_x - x)].y;
    one.m_p_uchar3[l_y * one.m_size.x + l_x].z = two.m_p_uchar4[(l_y - y) * two.m_size.x + (l_x - x)].z;
}

void cu_create_overlap( CudaImg one, CudaImg two, int x, int y )
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks( ( one.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( one.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_overlap<<< l_blocks, l_threads >>>( one, two, x, y );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// ==============================

__global__ void kernel_resize( CudaImg input, CudaImg output )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= output.m_size.y ) return;
    if ( l_x >= output.m_size.x ) return;

    //int xRatio = input.m_size.x / width;
    //int yRatio = input.m_size.y / height;

    //int pos = (l_y / yRatio) * output.m_size.x + (l_x / xRatio);

    //printf("%d\n", pos);

    //output.m_p_uchar4[pos].x = (output.m_p_uchar4[pos].x + input.m_p_uchar4[l_y * input.m_size.x + l_x].x) /2;
    //output.m_p_uchar4[pos].y = (output.m_p_uchar4[pos].y + input.m_p_uchar4[l_y * input.m_size.x + l_x].y) /2;
    //output.m_p_uchar4[pos].z = (output.m_p_uchar4[pos].z + input.m_p_uchar4[l_y * input.m_size.x + l_x].z) /2;
    //output.m_p_uchar4[pos].w = (output.m_p_uchar4[pos].w + input.m_p_uchar4[l_y * input.m_size.x + l_x].w) /2;
}

void cu_create_resize( CudaImg input, CudaImg output )
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks( ( output.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( output.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_resize<<< l_blocks, l_threads >>>( input, output );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}