#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

__global__ void kernel_mirror( CudaImg from, CudaImg to )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= from.m_size.y ) return;
    if ( l_x >= from.m_size.x ) return;

    to.m_p_uchar3[l_y * to.m_size.x + (to.m_size.x - l_x)] = from.m_p_uchar3[l_y * from.m_size.x + l_x];
}

void cu_create_mirror( CudaImg from, CudaImg to )
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks( ( from.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( from.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_mirror<<< l_blocks, l_threads >>>( from, to );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// ==============================

__global__ void kernel_dark( CudaImg from, CudaImg to )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= from.m_size.y ) return;
    if ( l_x >= from.m_size.x ) return;

    to.m_p_uchar3[l_y * to.m_size.x + l_x].x = from.m_p_uchar3[l_y * from.m_size.x + l_x].x * ((float) l_y/from.m_size.y);
    to.m_p_uchar3[l_y * to.m_size.x + l_x].y = from.m_p_uchar3[l_y * from.m_size.x + l_x].y * ((float) l_y/from.m_size.y);
    to.m_p_uchar3[l_y * to.m_size.x + l_x].z = from.m_p_uchar3[l_y * from.m_size.x + l_x].z * ((float) l_y/from.m_size.y);
}

void cu_create_dark( CudaImg from, CudaImg to )
{
    hipError_t l_cerr;

    int l_block_size = 32;
    dim3 l_blocks( ( from.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( from.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_dark<<< l_blocks, l_threads >>>( from, to );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}