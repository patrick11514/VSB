// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage.
// Global variables usage in threads, the use of printf.
//
// Every thread displays information of its position in block,
// position of block in grid and global position.
//
// ***********************************************************************


#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

// Demo kernel will display all global variables of grid organization.
// Warning! Function printf is available from compute capability 2.x
__global__ void thread_hierarchy()
{
    // Global variables
    // Grid dimension -             gridDim
    // Block position in grid -     blockIdx
    // Block dimension -            blockDim
    // Thread position in block -   threadIdx
    int l_x = threadIdx.x + blockIdx.x * blockDim.x;
    int l_y = threadIdx.y + blockIdx.y * blockDim.y;
    printf( "Block{%d,%d}[%d,%d] Thread{%d,%d}[%d,%d] [%d,%d]\n",
        gridDim.x, gridDim.y, blockIdx.x, blockIdx.y,
        blockDim.x, blockDim.y, threadIdx.x, threadIdx.y, l_x, l_y );
}

void cu_run_cuda( dim3 t_grid_size, dim3 t_block_size )
{
    hipError_t l_cerr;
    // Following command can increase internal buffer for printf function
    /*l_cerr = cudaDeviceSetLimit( cudaLimitPrintfFifoSize, required_size );
    if ( l_err != cudaSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, cudaGetErrorString( l_cerr ) );
    */
    // Thread creation from selected kernel:
    // first parameter dim3 is grid dimension
    // second parameter dim3 is block dimension
    thread_hierarchy<<< t_grid_size, t_block_size >>>();

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    // Output from printf is in GPU memory. 
    // To get its contens it is necessary to synchronize device.

    hipDeviceSynchronize();
}

